
#include <hip/hip_runtime.h>
// Cuckaroo Cycle, a memory-hard proof-of-work by John Tromp and team Grin
// Copyright (c) 2018 Jiri Photon Vadura and John Tromp
// Modified work Copyright (c) 2019 Lip Wee Yeo
// This Grinble miner file is covered by the FAIR MINING license

#ifdef __INTELLISENSE__

#ifndef __CUDACC__
#	define __CUDACC__
#endif // !__CUDACC__

#ifndef __CUDA_ARCH__
#	define __CUDA_ARCH__ 610
#endif // !__CUDA_ARCH__

#include <device_launch_parameters.h>
#include <device_atomic_functions.h>
#include <device_functions.h>
#include <sm_20_intrinsics.h>
#include <vector_functions.h>
#include <math_functions.h>
#include <host_defines.h>

#ifdef __constant__
#	undef __constant__
#endif // __constant__
#define __constant__

#ifdef __shared__
#	undef __shared__
#endif // __shared__
#define __shared__

#ifdef __global__
#	undef __global__
#endif // __global__
#define __global__

#ifdef __device__
#	undef __device__
#endif // __device__
#define __device__

#endif // __INTELLISENSE__

typedef unsigned char u8;
typedef unsigned short u16;
typedef unsigned int u32;
typedef unsigned long long u64;

#define DUCK_SIZE_A 129
#define DUCK_SIZE_B 82

#define DUCK_A_EDGES (DUCK_SIZE_A * 1024)
#define DUCK_A_EDGES_64 (DUCK_A_EDGES * 64)

#define DUCK_B_EDGES (DUCK_SIZE_B * 1024)
#define DUCK_B_EDGES_64 (DUCK_B_EDGES * 64)

#define EDGE_BLOCK_SIZE 64
#define EDGE_BLOCK_MASK (EDGE_BLOCK_SIZE - 1)

#define EDGEBITS 29
#define NEDGES (1u << EDGEBITS)
#define EDGEMASK (NEDGES - 1)

#define CTHREADS 512
#define BKTMASK4K (4096-1)
#define BKTGRAN 32

#define ROTL(x,b) ( ((x) << (b)) | ( (x) >> (64 - (b))) )

#define SIPROUND \
  { \
    v0 += v1; v2 += v3; v1 = ROTL(v1,13); \
    v3 = ROTL(v3,16); v1 ^= v0; v3 ^= v2; \
    v0 = ROTL(v0,32); v2 += v1; v0 += v3; \
    v1 = ROTL(v1,17); v3 = ROTL(v3,21); \
    v1 ^= v2; v3 ^= v0; v2 = ROTL(v2,32); \
  }

#define READ_2B_COUNTER(e,b) ( (e[(b >> 5) + 4096] & (1 << (b & 0x1f))) > 0 )

#define INCR_2B_COUNTER(e,b) \
  { \
    const int w = b >> 5; const u32 m = 1 << (b & 0x1f); \
    const u32 old  = atomicOr(ecounters + w, m) & m; \
    if (old > 0) atomicOr(e + w + 4096, m); \
  }

extern "C" {

	__constant__ u64 nonce[4];
	__constant__ u64 recovery[42];

	__global__ void FluffySeed2A(ulonglong4* buffer, int* indexes)
	{
		const u32 gid = blockDim.x * blockIdx.x + threadIdx.x;
		const u32 lid = threadIdx.x;

		__shared__ u64 tmp[64][16];
		__shared__ u32 counters[64];

		u64 sipblock[64];
		u64 v0, v1, v2, v3;

		if (lid < 64) counters[lid] = 0;

		__syncthreads();

		for (int i = 0; i < 1024 * 2; i += EDGE_BLOCK_SIZE)
		{
			const u64 blockNonce = gid * (1024 * 2) + i;

			v0 = nonce[0];
			v1 = nonce[1];
			v2 = nonce[2];
			v3 = nonce[3];

			for (int b = 0; b < EDGE_BLOCK_SIZE; ++b)
			{
				v3 ^= blockNonce + b;
				for (int r = 0; r < 2; ++r) SIPROUND;
				v0 ^= blockNonce + b;
				v2 ^= 0xff;
				for (int r = 0; r < 4; ++r) SIPROUND;
				sipblock[b] = (v0 ^ v1) ^ (v2 ^ v3);
			}

			const u64 last = sipblock[EDGE_BLOCK_MASK];

			for (int s = 0; s < EDGE_BLOCK_SIZE; ++s)
			{
				const u64 lookup = (s == EDGE_BLOCK_MASK) ? last : sipblock[s] ^ last;
				const uint2 hash = make_uint2(lookup & EDGEMASK, (lookup >> 32) & EDGEMASK);
				const int bucket = hash.x & 63;

				__syncthreads();

				const int counter = atomicAdd(counters + bucket, 1u);
				const int counterLocal = counter % 16;
				tmp[bucket][counterLocal] = hash.x | ((u64)hash.y << 32);

				__syncthreads();

				if ((counter > 0) && (counterLocal == 0 || counterLocal == 8))
				{
					const int cnt = min(atomicAdd(indexes + bucket, 8), (DUCK_A_EDGES_64 - 8));
					const int idx = (bucket * DUCK_A_EDGES_64 + cnt) / 4;

					buffer[idx] = make_ulonglong4(
						atomicExch(&tmp[bucket][8 - counterLocal], 0),
						atomicExch(&tmp[bucket][9 - counterLocal], 0),
						atomicExch(&tmp[bucket][10 - counterLocal], 0),
						atomicExch(&tmp[bucket][11 - counterLocal], 0)
					);
					buffer[idx + 1] = make_ulonglong4(
						atomicExch(&tmp[bucket][12 - counterLocal], 0),
						atomicExch(&tmp[bucket][13 - counterLocal], 0),
						atomicExch(&tmp[bucket][14 - counterLocal], 0),
						atomicExch(&tmp[bucket][15 - counterLocal], 0)
					);
				}
			}
		}

		__syncthreads();

		if (lid < 64)
		{
			const int counterBase = (counters[lid] % 16) >= 8 ? 8 : 0;
			const int cnt = min(atomicAdd(indexes + lid, 8), (DUCK_A_EDGES_64 - 8));
			const int idx = (lid * DUCK_A_EDGES_64 + cnt) / 4;
			buffer[idx] = make_ulonglong4(tmp[lid][counterBase], tmp[lid][counterBase + 1], tmp[lid][counterBase + 2], tmp[lid][counterBase + 3]);
			buffer[idx + 1] = make_ulonglong4(tmp[lid][counterBase + 4], tmp[lid][counterBase + 5], tmp[lid][counterBase + 6], tmp[lid][counterBase + 7]);
		}
	}

	__global__ void FluffySeed2B(const uint2* source, ulonglong4* destination, const int* sourceIndexes, int* destinationIndexes, int startBlock)
	{
		const u32 lid = threadIdx.x;
		const u32 group = blockIdx.x;

		__shared__ u64 tmp[64][16];
		__shared__ int counters[64];

		if (lid < 64) counters[lid] = 0;

		__syncthreads();

		const int offsetMem = startBlock * DUCK_A_EDGES_64;
		const int myBucket = group / BKTGRAN;
		const int microBlockNo = group % BKTGRAN;
		const int bucketEdges = min(sourceIndexes[myBucket + startBlock], (DUCK_A_EDGES_64));
		const int microBlockEdgesCount = (DUCK_A_EDGES_64 / BKTGRAN);
		const int loops = (microBlockEdgesCount / 128);

		for (int i = 0; i < loops; ++i)
		{
			const int edgeIndex = (microBlockNo * microBlockEdgesCount) + (128 * i) + lid;
			const uint2 edge = source[offsetMem + (myBucket * DUCK_A_EDGES_64) + edgeIndex];
			const bool skip = (edgeIndex >= bucketEdges) || (edge.x == 0 && edge.y == 0);
			const int bucket = (edge.x >> 6) & (64 - 1);

			__syncthreads();

			const int counter = skip ? 0 : atomicAdd(counters + bucket, 1u);
			const int counterLocal = skip ? 0 : counter % 16;
			tmp[bucket][counterLocal] = edge.x | ((u64)edge.y << 32);

			__syncthreads();

			if ((counter > 0) && (counterLocal == 0 || counterLocal == 8))
			{
				const int cnt = min(atomicAdd(destinationIndexes + startBlock * 64 + myBucket * 64 + bucket, 8), (DUCK_A_EDGES - 8));
				const int idx = ((myBucket * 64 + bucket) * DUCK_A_EDGES + cnt) / 4;

				destination[idx] = make_ulonglong4(
					atomicExch(&tmp[bucket][8 - counterLocal], 0),
					atomicExch(&tmp[bucket][9 - counterLocal], 0),
					atomicExch(&tmp[bucket][10 - counterLocal], 0),
					atomicExch(&tmp[bucket][11 - counterLocal], 0)
				);
				destination[idx + 1] = make_ulonglong4(
					atomicExch(&tmp[bucket][12 - counterLocal], 0),
					atomicExch(&tmp[bucket][13 - counterLocal], 0),
					atomicExch(&tmp[bucket][14 - counterLocal], 0),
					atomicExch(&tmp[bucket][15 - counterLocal], 0)
				);
			}
		}

		__syncthreads();

		if (lid < 64)
		{
			const int counterBase = (counters[lid] % 16) >= 8 ? 8 : 0;
			const int cnt = min(atomicAdd(destinationIndexes + startBlock * 64 + myBucket * 64 + lid, 8), (DUCK_A_EDGES - 8));
			const int idx = ((myBucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4;
			destination[idx] = make_ulonglong4(tmp[lid][counterBase], tmp[lid][counterBase + 1], tmp[lid][counterBase + 2], tmp[lid][counterBase + 3]);
			destination[idx + 1] = make_ulonglong4(tmp[lid][counterBase + 4], tmp[lid][counterBase + 5], tmp[lid][counterBase + 6], tmp[lid][counterBase + 7]);
		}
	}

	__global__ void FluffyRound(const uint2* source, uint2* destination, const int* sourceIndexes, int* destinationIndexes, const int bktInSize, const int bktOutSize)
	{
		const u32 lid = threadIdx.x;
		const u32 group = blockIdx.x;

		__shared__ u32 ecounters[8192];

		const int bktInGroupSize = bktInSize * group;
		const int edgesInBucket = min(sourceIndexes[group], bktInSize);
		const int loops = (edgesInBucket + CTHREADS) / CTHREADS;

		for (int i = 0; i < 16; ++i)
			ecounters[lid + (512 * i)] = 0;

		__syncthreads();

		for (int i = 0; i < loops; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucket)
			{
				const uint2 edge = __ldg(&source[bktInGroupSize + lindex]);

				if (edge.x > 0 || edge.y > 0)
					INCR_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12);
			}
		}

		__syncthreads();

		for (int i = 0; i < loops; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucket)
			{
				const uint2 edge = __ldg(&source[bktInGroupSize + lindex]);

				if (edge.x > 0 || edge.y > 0)
					if (READ_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12))
					{
						const int bucket = edge.y & BKTMASK4K;
						const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
						destination[(bucket * bktOutSize) + bktIdx] = make_uint2(edge.y, edge.x);
					}
			}
		}
	}

	__global__ void FluffyRound_J(const uint2* sourceA, const uint2* sourceB, uint2* destination, const int* sourceIndexes, int* destinationIndexes, const int bktInSize, const int bktOutSize)
	{
		const u32 lid = threadIdx.x;
		const u32 group = blockIdx.x;

		__shared__ u32 ecounters[8192];

		const int bktInGroupSize = bktInSize * group;
		const int edgesInBucketA = min(sourceIndexes[group], bktInSize);
		const int edgesInBucketB = min(sourceIndexes[group + 4096], bktInSize);

		const int loopsA = (edgesInBucketA + CTHREADS) / CTHREADS;
		const int loopsB = (edgesInBucketB + CTHREADS) / CTHREADS;

		for (int i = 0; i < 16; ++i)
			ecounters[lid + (512 * i)] = 0;

		__syncthreads();

		for (int i = 0; i < loopsA; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucketA)
			{
				const uint2 edge = sourceA[bktInGroupSize + lindex];

				if (edge.x > 0 || edge.y > 0)
					INCR_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12);
			}
		}

		for (int i = 0; i < loopsB; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucketB)
			{
				const uint2 edge = sourceB[bktInGroupSize + lindex];

				if (edge.x > 0 || edge.y > 0)
					INCR_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12);
			}
		}

		__syncthreads();

		for (int i = 0; i < loopsA; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucketA)
			{
				const uint2 edge = sourceA[bktInGroupSize + lindex];

				if (edge.x > 0 || edge.y > 0)
					if (READ_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12))
					{
						const int bucket = edge.y & BKTMASK4K;
						const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
						destination[(bucket * bktOutSize) + bktIdx] = make_uint2(edge.y, edge.x);
					}
			}
		}

		for (int i = 0; i < loopsB; ++i)
		{
			const int lindex = (i * CTHREADS) + lid;

			if (lindex < edgesInBucketB)
			{
				const uint2 edge = sourceB[bktInGroupSize + lindex];

				if (edge.x > 0 || edge.y > 0)
					if (READ_2B_COUNTER(ecounters, (edge.x & EDGEMASK) >> 12))
					{
						const int bucket = edge.y & BKTMASK4K;
						const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
						destination[(bucket * bktOutSize) + bktIdx] = make_uint2(edge.y, edge.x);
					}
			}
		}
	}

	__global__ void FluffyTail(const uint2* source, uint2* destination, const int* sourceIndexes, int* destinationIndexes)
	{
		const u32 lid = threadIdx.x;
		const u32 group = blockIdx.x;

		const int myEdges = sourceIndexes[group];
		__shared__ int destIdx;

		if (lid == 0) destIdx = atomicAdd(destinationIndexes, myEdges);

		__syncthreads();

		if (lid < myEdges) destination[destIdx + lid] = source[group * DUCK_B_EDGES / 4 + lid];
	}

	__global__ void FluffyRecovery(int* indexes)
	{
		const u32 gid = blockDim.x * blockIdx.x + threadIdx.x;
		const u32 lid = threadIdx.x;

		__shared__ u32 nonces[42];

		u64 sipblock[64];
		u64 v0, v1, v2, v3;

		if (lid < 42) nonces[lid] = 0;

		__syncthreads();

		for (int i = 0; i < 1024; i += EDGE_BLOCK_SIZE)
		{
			const u64 blockNonce = gid * 1024 + i;

			v0 = nonce[0];
			v1 = nonce[1];
			v2 = nonce[2];
			v3 = nonce[3];

			for (int b = 0; b < EDGE_BLOCK_SIZE; ++b)
			{
				v3 ^= blockNonce + b;
				for (int r = 0; r < 2; ++r) SIPROUND;
				v0 ^= blockNonce + b;
				v2 ^= 0xff;
				for (int r = 0; r < 4; ++r) SIPROUND;
				sipblock[b] = (v0 ^ v1) ^ (v2 ^ v3);
			}

			const u64 last = sipblock[EDGE_BLOCK_MASK];

			for (int s = EDGE_BLOCK_MASK; s >= 0; --s)
			{
				const u64 lookup = (s == EDGE_BLOCK_MASK) ? last : sipblock[s] ^ last;
				const u64 u = lookup & EDGEMASK;
				const u64 v = (lookup >> 32) & EDGEMASK;
				const u64 a = u | (v << 32);
				const u64 b = v | (u << 32);

				for (int i = 0; i < 42; ++i)
					if ((recovery[i] == a) || (recovery[i] == b))
						nonces[i] = blockNonce + s;
			}
		}

		__syncthreads();

		if (lid < 42)
			if (nonces[lid] > 0)
				indexes[lid] = nonces[lid];
	}
}
